/*
 * vecAdd.cu
 *
 *  Created on: Jul 31, 2015
 *      Author: lsa
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vecAdd(int *a, int *b, int *c){
	*c = *a + *b;
}

int main(){
	int a, b, c;					//Host side variables
	int *d_a, *d_b, *d_c;		//Device side variables
	int size = sizeof(int);

	//Allocate device copies of a, b and c
	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);

	//Allocate host side arrays
	a = 2;
	b = 7;

	//Copy inputs to device
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

	//Launch kernel
	vecAdd<<<1,1>>>(d_a, d_b, d_c);

	//Copy result back to host
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

	//Cleanup
	hipFree(d_a); hipFree(d_b); hipFree(d_c);

	//Print
	printf("\nGPU result: %d\nCPU result: %d", c, a+b);

	return 0;
}
